#include "hip/hip_runtime.h"
// tfidf_gpu.cu
#include "preprocess.hpp"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void compute_tf_df(
    const uint32_t *docs, const int *doc_offsets, int num_docs,
    uint32_t *tf_counts, uint32_t *df_counts, int vocab_size)
{
    int doc_id = blockIdx.x;
    if (doc_id >= num_docs) return;

    int start = doc_offsets[doc_id];
    int end   = doc_offsets[doc_id + 1];

    extern __shared__ uint8_t term_flags[];
    for (int i = threadIdx.x; i < vocab_size; i += blockDim.x)
        term_flags[i] = 0;
    __syncthreads();

    for (int i = start + threadIdx.x; i < end; i += blockDim.x) {
        uint32_t term = docs[i];
        if (term < vocab_size) {
            atomicAdd(&tf_counts[doc_id * vocab_size + term], 1);
            term_flags[term] = 1;
        }
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        for (int t = 0; t < vocab_size; ++t)
            if (term_flags[t]) atomicAdd(&df_counts[t], 1);
    }
}

__global__ void compute_tfidf(
    const uint32_t *tf_counts, const uint32_t *df_counts,
    float *tfidf, int num_docs, int vocab_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = num_docs * vocab_size;
    if (idx >= total) return;

    int term_id = idx % vocab_size;
    int df = df_counts[term_id];
    if (df == 0) return;

    float tf = static_cast<float>(tf_counts[idx]);
    float idf = logf(static_cast<float>(num_docs) / df);
    tfidf[idx] = tf * idf;
}

// -------- GPU runner --------

void run_tfidf_on_gpu(const CorpusData &data) {
    int num_docs = data.num_docs;
    int vocab_size = data.vocab_size;
    const auto &flat_docs = data.flat_docs;
    const auto &doc_offsets = data.doc_offsets;

    cout << "Launching GPU kernels... (docs=" << num_docs
              << ", vocab=" << vocab_size << ")\n";

    uint32_t *d_docs, *d_tf, *d_df;
    int *d_offsets;
    float *d_tfidf;

    hipMalloc(&d_docs, flat_docs.size() * sizeof(uint32_t));
    hipMalloc(&d_offsets, doc_offsets.size() * sizeof(int));
    hipMalloc(&d_tf, num_docs * vocab_size * sizeof(uint32_t));
    hipMalloc(&d_df, vocab_size * sizeof(uint32_t));
    hipMalloc(&d_tfidf, num_docs * vocab_size * sizeof(float));

    hipMemcpy(d_docs, flat_docs.data(), flat_docs.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, doc_offsets.data(), doc_offsets.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_tf, 0, num_docs * vocab_size * sizeof(uint32_t));
    hipMemset(d_df, 0, vocab_size * sizeof(uint32_t));

    compute_tf_df<<<num_docs, 128, vocab_size * sizeof(uint8_t)>>>(
        d_docs, d_offsets, num_docs, d_tf, d_df, vocab_size);
    hipDeviceSynchronize();

    int threads = 128;
    int blocks = (num_docs * vocab_size + threads - 1) / threads;
    compute_tfidf<<<blocks, threads>>>(d_tf, d_df, d_tfidf, num_docs, vocab_size);
    hipDeviceSynchronize();

    vector<float> h_tfidf(num_docs * vocab_size);
    hipMemcpy(h_tfidf.data(), d_tfidf, h_tfidf.size() * sizeof(float), hipMemcpyDeviceToHost);

    cout << "\n=== TF-IDF Matrix (first 10 terms per doc) ===\n";
    for (int d = 0; d < num_docs; ++d) {
        cout << "Doc " << d << ": [ ";
        for (int t = 0; t < min(10, vocab_size); ++t)
            cout << h_tfidf[d * vocab_size + t] << " ";
        cout << "... ]\n";
    }

    hipFree(d_docs);
    hipFree(d_offsets);
    hipFree(d_tf);
    hipFree(d_df);
    hipFree(d_tfidf);
}

// -------- Main entry --------
int main() {
    cout << "Enter corpus folder path: ";
    string folder;
    getline(cin, folder);

    try {
        CorpusData data = preprocess_corpus(folder);
        run_tfidf_on_gpu(data);
    } catch (const std::exception &e) {
        cerr << "Error: " << e.what() << "\n";
        return 1;
    }

    cout << "\nTF-IDF complete.\n";
    return 0;
}
